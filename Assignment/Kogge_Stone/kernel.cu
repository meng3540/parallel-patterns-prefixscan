#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 1024
#define MAX 9
#define P 12


__global__ void scan2(int* X, int* Y, int InputSize) {
    __shared__ int XY[N];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < InputSize) {
        XY[threadIdx.x] = X[i];
    }
    for (int j = 1; j < blockDim.x; j *= 2) {
        __syncthreads();
        if (threadIdx.x >= j) {
            XY[threadIdx.x] += XY[threadIdx.x - j];
        }
        Y[i] = XY[threadIdx.x];
    }
}


int main(int argc, char** argv) {

    int* data, * h_obstacle, * h_prefixSum; //Host pointers
    int* d_obstacle, * d_prefixSum2; //Device pointers

    //Allocate and initialize matrices
    data = (int*)malloc(N * sizeof(int));
    h_obstacle = (int*)malloc(N * sizeof(int));
    h_prefixSum = (int*)malloc(N * sizeof(int));

    //Populate with distances
    for (int i = 0; i < N; i++) {
        int j = 8;
        if (i % j == 0) {
            data[i] = 3;
            //if (i+1 <N) { //this is used to make the obstacles wider in the array.
                //data[i+1]=3;
                //i++;
            //}
        }
        else {
            data[i] = 9;
        }
    }

    //Check if obstacle detected
    for (int i = 0; i < N; i++) {
        if (data[i] < MAX) {
            h_obstacle[i] = 1;
        }
        else {
            h_obstacle[i] = 0;
        }
    }

    //@@Display result
    printf("Array Size = %d\n", N);
    printf("\nData Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%d  ", data[i]);
    }
    printf(" . . . ");
    for (int i = N - P; i < N; i++) {
        printf("  %d", data[i]);
    }

    printf("\nObstacle Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%d  ", h_obstacle[i]);
    }
    printf(" . . . ");
    for (int i = N - P; i < N; i++) {
        printf("  %d", h_obstacle[i]);
    }



    //@@Allocate GPU Memory
    hipMalloc((void**)&d_obstacle, N * sizeof(int));

    hipMalloc((void**)&d_prefixSum2, N * sizeof(int));


    //@@Copy memory to GPU
    hipMemcpy(d_obstacle, h_obstacle, N * sizeof(int), hipMemcpyHostToDevice);

    //@@Initialize the grid and block dimensions here
    dim3 blockSize(N);
    dim3 gridSize((int)ceil((float)N / blockSize.x));

    hipEvent_t start2, stop2;

    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    float t2 = 0;

    //@@Launch GPU kernel
    hipEventRecord(start2);
    scan2 << <gridSize, blockSize >> > (d_obstacle, d_prefixSum2, N);
    hipDeviceSynchronize();
    hipEventRecord(stop2);

    //@@Copy GPU memory back to CPU
    hipMemcpy(h_prefixSum, d_prefixSum2, N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop2);
    hipEventElapsedTime(&t2, start2, stop2);
    printf("\n\nTime (Kogge-Stone): %f ms", t2);

    printf("\nPrefix Sum Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%3.0d", h_prefixSum[i]);
    }
    printf("  . . . ");
    for (int i = N - P; i < N; i++) {
        printf("%3.0d", h_prefixSum[i]);
    }
    printf("\n%d obstacles detected", h_prefixSum[N - 1]);

    //@@Free GPU memory
    hipFree(d_obstacle);
    hipFree(d_prefixSum2);

    //@@Free host memory
    free(h_obstacle);
    free(h_prefixSum);
}
