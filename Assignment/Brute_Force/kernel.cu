#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 1024
#define MAX 9
#define P 12


__global__ void scan1(int* X, int* Y, int InputSize) {
    __shared__ int XY[N];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Y[i] = 0;
    for (int j = 0; j <= i; j++) {
        Y[i] += X[j];
    }
}


int main(int argc, char** argv) {

    int* data, * h_obstacle, * h_prefixSum; //Host pointers
    int* d_obstacle, * d_prefixSum1; //Device pointers

    //Allocate and initialize matrices
    data = (int*)malloc(N * sizeof(int));
    h_obstacle = (int*)malloc(N * sizeof(int));
    h_prefixSum = (int*)malloc(N * sizeof(int));

    //Populate with distances
    for (int i = 0; i < N; i++) {
        int j = 8;
        if (i % j == 0) {
            data[i] = 3;
            //if (i+1 <N) { //this is used to make the obstacles wider in the array.
                //data[i+1]=3;
                //i++;
            //}
        }
        else {
            data[i] = 9;
        }
    }

    //Check if obstacle detected
    for (int i = 0; i < N; i++) {
        if (data[i] < MAX) {
            h_obstacle[i] = 1;
        }
        else {
            h_obstacle[i] = 0;
        }
    }

    //@@Display result
    printf("Array Size = %d\n", N);
    printf("\nData Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%d  ", data[i]);
    }
    printf(" . . . ");
    for (int i = N - P; i < N; i++) {
        printf("  %d", data[i]);
    }

    printf("\nObstacle Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%d  ", h_obstacle[i]);
    }
    printf(" . . . ");
    for (int i = N - P; i < N; i++) {
        printf("  %d", h_obstacle[i]);
    }



    //@@Allocate GPU Memory
    hipMalloc((void**)&d_obstacle, N * sizeof(int));
    hipMalloc((void**)&d_prefixSum1, N * sizeof(int));

    //@@Copy memory to GPU
    hipMemcpy(d_obstacle, h_obstacle, N * sizeof(int), hipMemcpyHostToDevice);

    //@@Initialize the grid and block dimensions here
    dim3 blockSize(N);
    dim3 gridSize((int)ceil((float)N / blockSize.x));

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    float t1 = 0;


    //@@Launch GPU kernel
    hipEventRecord(start1);
    scan1 << <gridSize, blockSize >> > (d_obstacle, d_prefixSum1, N);
    hipDeviceSynchronize();
    hipEventRecord(stop1);

    //@@Copy GPU memory back to CPU
    hipMemcpy(h_prefixSum, d_prefixSum1, N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop1);
    hipEventElapsedTime(&t1, start1, stop1);
    printf("\n\nTime (Brute-Force): %f ms", t1);

    printf("\nPrefix Sum Array:\n");
    for (int i = 0; i < P; i++) {
        printf("%3.0d", h_prefixSum[i]);
    }
    printf("  . . . ");
    for (int i = N - P; i < N; i++) {
        printf("%3.0d", h_prefixSum[i]);
    }
    printf("\n%3.0d obstacles detected", h_prefixSum[N - 1]);


    //@@Free GPU memory
    hipFree(d_obstacle);
    hipFree(d_prefixSum1);

    //@@Free host memory
    free(h_obstacle);
    free(h_prefixSum);
}
